#include "hip/hip_runtime.h"
//1NN classifier based on output of Propagating_1NN

#include "utilities.h"
#include "calcUtilities.h"
#include "distUtilities.h"
#include "evaluationUtilities.h"
#include "hip/hip_runtime.h"
#include ""
#include <io.h>
#include <direct.h>
#include <string>

#define INF 1e6
#define EPS 1e-16
#define MAX_CHAR 10

void getDists_DTW_D(double *distMtx_DTW_D, double *distMtx_ED, double *distMtx_DTW, int numTrain, int numTest) {
	for (int i = 0; i < numTest; i++) {
		for (int j = 0; j < numTrain; j++) {
			distMtx_DTW_D[i * numTrain + j] = distMtx_DTW[i * numTrain + j] / (distMtx_ED[i * numTrain + j] + EPS);
		}
	}
}

void classifyNN(int *preLabels, int *groundTruthLabels, double *distMtx, int numTrain, int numTest) {
	double minDist;
	int nnInd;
	for (int i = 0; i < numTest; i++) {
		min(minDist, nnInd, distMtx + i * numTrain, numTrain);
		preLabels[i] = groundTruthLabels[nnInd];
	}
}

int main(int argc, char **argv) {

	//parameter settings
	if (argc < 8)
		exit(1);
	std::string datasetName = argv[1];
	const int numTrain = atoi(argv[2]);
	const int numP = atoi(argv[3]);
	const int numPLabeled = atoi(argv[4]);
	const int tsLen = atoi(argv[5]);
	const int numTest = atoi(argv[6]);
	const int seedId = atoi(argv[7]);
	const int minNumIters = argc > 8 ? atoi(argv[8]) : 5;
	const int maxNumIters = argc > 9 ? atoi(argv[9]) : numTrain * 2 / 3 - numPLabeled;
	const int maxThreadsPerBlock = argc > 10 ? atoi(argv[10]) : 8;
	const int maxBlocksPerGrid = argc > 11 ? atoi(argv[11]) : 8;
	const std::string path = argc > 12 ? argv[12] : "..\\sample_data\\" + datasetName;

	/*std::string datasetName = "Car";
	const int numTrain = 60;
	const int numP = 16;
	const int numPLabeled = 2;
	const int tsLen = 577;
	const int numTest = 60;
	const int seedId = 0;
	const int minNumIters = 5;
	const int maxNumIters = numTrain * 2 / 3 - numPLabeled;
	const int maxThreadsPerBlock = 8;
	const int maxBlocksPerGrid = 8;
	const std::string path = "..\\sample_data\\" + datasetName;*/

	const int numSeeds = numP < 10 ? numP : 10;
	if (seedId >= numSeeds)
		exit(1);

	//training data
	long long trainTssBytes = numTrain * tsLen * sizeof(double);
	double *trainTss = (double*)malloc(trainTssBytes);
	long long trainLabelsBytes = numTrain * sizeof(int);
	int *trainLabels = (int*)malloc(trainLabelsBytes);
	importTimeSeries(trainTss, trainLabels, path, datasetName, "TRAIN", numTrain, tsLen);
	relabel(trainLabels, numTrain, 1);

	//testing data
	long long testTssBytes = numTest * tsLen * sizeof(double);
	double *testTss = (double*)malloc(testTssBytes);
	long long testLabelsBytes = numTest * sizeof(int);
	int *testLabels = (int*)malloc(testLabelsBytes);
	importTimeSeries(testTss, testLabels, path, datasetName, "TEST", numTest, tsLen);
	relabel(testLabels, numTest, 1);

	char s_seedId[MAX_CHAR], s_minNumIters[MAX_CHAR], s_maxNumIters[MAX_CHAR];
	_itoa(seedId, s_seedId, 10);
	_itoa(minNumIters, s_minNumIters, 10);
	_itoa(maxNumIters, s_maxNumIters, 10);

	//warp and transPreLabels
	int *warpAndLabels = (int *)malloc(27 * (numTrain + 1) * sizeof(int));
	std::string fName = path + "\\" + datasetName + "_P1NN_warps_and_preLabels_" + s_seedId + "_" + s_minNumIters + "_" + s_maxNumIters + ".txt";
	importMatrix(warpAndLabels, fName, 27, numTrain + 1, 1);

	//ED distMtx
	double *trainTss_in;
	hipMalloc(&trainTss_in, trainTssBytes);
	hipMemcpy(trainTss_in, trainTss, trainTssBytes, hipMemcpyHostToDevice);
	double *testTss_in;
	hipMalloc(&testTss_in, testTssBytes);
	hipMemcpy(testTss_in, testTss, testTssBytes, hipMemcpyHostToDevice);
	double *dists_out;
	long long pDistsBytes = numTest * numTrain * sizeof(double);
	hipMalloc(&dists_out, pDistsBytes);
	int blockSize = numTest < maxThreadsPerBlock ? numTest : maxThreadsPerBlock;
	int gridSize = ceil((double)numTest / blockSize) < maxBlocksPerGrid ? ceil((double)numTest / blockSize) : maxBlocksPerGrid;
	getPDists_DTW << <gridSize, blockSize >> > (trainTss_in, testTss_in, dists_out, numTrain, numTest, tsLen, 0);	//ED is DTW with zero warp.
	double *distMtx_ED = (double *)malloc(pDistsBytes);
	hipMemcpy(distMtx_ED, dists_out, pDistsBytes, hipMemcpyDeviceToHost);

	double warp, p, r, f;
	int intWarp;
	int *transPreLabels, *inducPreLabels = (int *)malloc(numTest * sizeof(int));
	double *inducFscores = (double *)malloc(27 * sizeof(double));
	double *distMtx, *distMtx_DTW = (double *)malloc(pDistsBytes);
	char s_warp[MAX_CHAR];
	for (int i = 0; i < 27; i++) {
		intWarp = warpAndLabels[i * (numTrain + 1)];
		if (intWarp < 0) {
			inducFscores[i] = -1;
			continue;
		}
		transPreLabels = warpAndLabels + i * (numTrain + 1) + 1;

		if (i < 9)	//ED
			distMtx = distMtx_ED;
		else {
			warp = (double)intWarp / 100;
			getPDists_DTW << <gridSize, blockSize >> > (trainTss_in, testTss_in, dists_out, numTrain, numTest, tsLen, warp);	//ED is DTW with zero warp.
			hipMemcpy(distMtx_DTW, dists_out, pDistsBytes, hipMemcpyDeviceToHost);

			if (i < 18) {	//DTW
				distMtx = distMtx_DTW;
			}
			else {	//DTW-D
				getDists_DTW_D(distMtx, distMtx_ED, distMtx_DTW, numTrain, numTest);
			}
		}

		classifyNN(inducPreLabels, transPreLabels, distMtx, numTrain, numTest);
		prf(p, r, f, testLabels, inducPreLabels, numTest);
		inducFscores[i] = f;

	}

	std::string methods[]{"ED_oracle", "ED_WK", "ED_RW", "ED_BHRK", "ED_GBTRM_1", "ED_GBTRM_2", "ED_GBTRM_3", "ED_GBTRM_4", "ED_GBTRM_5", 
		"DTW_oracle", "DTW_WK", "DTW_RW", "DTW_BHRK", "DTW_GBTRM_1", "DTW_GBTRM_2", "DTW_GBTRM_3", "DTW_GBTRM_4", "DTW_GBTRM_5", 
		"DTWD_oracle", "DTWD_WK", "DTWD_RW", "DTWD_BHRK", "DTWD_GBTRM_1", "DTWD_GBTRM_2", "DTWD_GBTRM_3", "DTWD_GBTRM_4", "DTWD_GBTRM_5"};

	fName = path + "\\" + datasetName + "_P1NN_test_fscores_" + s_seedId + "_" + s_minNumIters + "_" + s_maxNumIters + ".txt";
	std::ofstream fout;
	fout.open(fName);
	for (int i = 0; i < 27; i++) {
		std::cout << methods[i] << ": " << inducFscores[i] << std::endl;
		fout << methods[i] << ": " << inducFscores[i] << std::endl;
	}
	fout.close();
	std::cout << std::endl;

	hipFree(trainTss_in);
	hipFree(testTss_in);
	hipFree(dists_out);
	free(trainTss);
	free(trainLabels);
	free(testTss);
	free(testLabels);
	free(warpAndLabels);
	free(inducPreLabels);
	free(inducFscores);
	free(distMtx_ED);
	free(distMtx_DTW);

	return 0;
}